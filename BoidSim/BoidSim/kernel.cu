#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdint.h>
#include "boid.h"
#include "hipcub/hipcub.hpp"
#include "cub/device/device_radix_sort.cuh"

#include <stdio.h>
#include <algorithm>
#include "cub/util_allocator.cuh"
#include "cub/device/device_radix_sort.cuh"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"

// includes, cuda
#include <windows.h>
//#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

/* 
Compile on Linux machines in NC:  
/chalmers/sw/sup64/cuda_toolkit-9.0.176.4/bin/nvcc --dont-use-profile -ldir /chalmers/sw/sup64/cuda_toolkit-9.0.176.4/nvvm/libdevice/ -I /chalmers/sw/sup64/cuda_toolkit-9.0.176.4/include -m64 -L /chalmers/sw/sup64/cuda_toolkit-9.0.176.4/lib64 ~/kernel.cu

You might need this before compiling: 
PATH=$PATH:/chalmers/sw/sup64/cuda_toolkit-9.0.176.4/nvvm/bin
PATH=$PATH:/chalmers/sw/sup64/cuda_toolkit-9.0.176.4/bin

There are many optimization flags. See here:
https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#precision-related-compiler-flags

*/ 

/* A useful macro for displaying CUDA errors */ 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace hipcub;

// Boid attributes
#define MAX_SPEED 30.0f
#define MIN_SPEED 20.0f // TODO 

// #define DEBUG

// These arrays hold the boids
extern Boid* boids;
Boid* boidsSorted = NULL;

// These arrays hold the (Z-order/morton encoded) cell ids
uint64_t* boidCellIDs = NULL;
uint64_t* boidCellIDsAlt = NULL;

// Array with all the boids. boidsSorted is a alternate array needed for the radixSort
int* boidIDs = NULL;
int* boidIDsAlt = NULL;

// Doublebuffers containing boidIDs and cellIDs, these are used by the radix sort function
DoubleBuffer<uint64_t> boidCellIDsBuf;
DoubleBuffer<int> boidIDsBuf;

// Calculate the maximum value of Morton encoded (Z-ordered) cell ids
#define shiftBitK(x, k) (int) ((x&(1<<k)) << k*2+2 | (x&(1<<k)) << k*2+1 | (x&(1<<k)) << k*2)
const int MAX_CELL_INDEX = (int) MAX_COORD/CELL_SIZE;
const int NR_CELLS = shiftBitK(MAX_CELL_INDEX, 10) 
                     |shiftBitK(MAX_CELL_INDEX, 9)
                     |shiftBitK(MAX_CELL_INDEX, 8)
                     |shiftBitK(MAX_CELL_INDEX, 7)
                     |shiftBitK(MAX_CELL_INDEX, 6)
                     |shiftBitK(MAX_CELL_INDEX, 5)
                     |shiftBitK(MAX_CELL_INDEX, 4)
                     |shiftBitK(MAX_CELL_INDEX, 3)
                     |shiftBitK(MAX_CELL_INDEX, 2)
                     |shiftBitK(MAX_CELL_INDEX, 1)
                     |shiftBitK(MAX_CELL_INDEX, 0);

// These parameters are used by the CUDA functions
int blockSize = 256;
int numBlocksBoids = (NR_BOIDS + blockSize - 1) / blockSize;
int numBlocksCells = (NR_CELLS + blockSize - 1) / blockSize;

// A tempory storage for new velocities allows parallel processing of the boids velocities 
glm::vec3* newVelocities;

// These arrays hold the start and end indices for each cell which contains boids
int* cellStartIndex;
int* cellEndIndex;

// Get the cell based on the boids position
inline __device__ glm::vec3 getCell(glm::vec3 pos){
    return glm::floor(pos * (1.0f/CELL_SIZE));
}

// The boids are given an ID, which is the index it has in the intial boid array
__global__ void initBoidIDs(int BoidIDs[], int nrBoids){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x; 
    if(i >= nrBoids) return; 
    BoidIDs[i] = i;
}

// Spreads out bits with two zeroes in between
// TODO: find the source and give credit
__device__ uint64_t spreadOutByThree(uint64_t i){
    i = (i | (i << 16)) & 0x030000FF;
    i = (i | (i <<  8)) & 0x0300F00F;
    i = (i | (i <<  4)) & 0x030C30C3;
    i = (i | (i <<  2)) & 0x09249249;
    return i;
}


// Hash cell coords to morton code with "magic numbers"
__device__ uint64_t bitInterlaceMagic(int x, int y, int z){
    return spreadOutByThree((uint64_t)x) | (spreadOutByThree((uint64_t)y) << 1) | (spreadOutByThree((uint64_t)z) << 2);
}

// This function is used when scanning the sorted boids cell-ids to see were cells starts and ends 
// dummyHostIndex is only for debugging on machines without device enabled
__global__ void detectCellIndexChange(int cellStarts[], int cellEnds[], uint64_t cellIDs[], int nrBoids){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x; 
    if(i >= nrBoids) return; 
    
    int cellID = cellIDs[i];
    int nextCellID = cellIDs[i+1];
    #ifdef DEBUG
    printf("Checking index %d which has value %d \n", i, cellID);
    #endif
    // TODO: determine if these if/else statements causes thread branching -> worse performance
    if(i == 0){ 
        // This is the case for the first element in the boid array 
        cellStarts[cellID] = i; 
        #ifdef DEBUG
        printf("Cell start was detected: from %d \n", cellID);
        #endif
    } else if (i == nrBoids - 1){ 
        // This is the case for the last element in the boid array
        cellEnds[cellID] = i;
        #ifdef DEBUG
        printf("Cell end was detected: from %d \n", cellID);
        #endif
        return;
    } 
    if (cellID != nextCellID){
        // A change in cell index was detected!
        #ifdef DEBUG
        printf("A cell change was detected: from %d to %d \n", cellID, nextCellID);
        #endif
        cellStarts[nextCellID] = i + 1;
        cellEnds[cellID] = i;
    }
}

// Update boid with index n
// WARNING! VERY MUCH TODO: RIGHT NOW IT CHECKS OUTSIDE WORLD BOUNDARIES FOR BOIDS
// IN EDGE CELLS, THIS WILL CAUSE ARRAY OUT OF-BOUNDS EXCEPTIONS
__global__ void computeVelocities(Boid boids[], int cellStarts[], int cellEnds[], uint64_t cellIDs[]
                                , int nrBoids, glm::vec3 newVelocities[]){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if(i >= nrBoids) return;
    int neighbourCount = 0;
    Boid b = boids[i]; // current boid whose neighbours we're checking
    // initialize default values for each rule
    glm::vec3 alignment = b.velocity;
	glm::vec3 separation = glm::vec3(0.0);
    glm::vec3 cohesion = glm::vec3(0.0);
    // Decide which cell current boid is in
    glm::vec3 cell = getCell(b.position);
    // Start checking all 27 neighbouring cells
    // TODO: find out a clever way to iterate over cells in order of the morton code to get 
    // more coherent memory accesses
    for(int i = -1; i <= 1; i++){
        for(int j = -1; j <= 1; j++){
            for(int k = -1; k <= 1; k++){
                // calculate the (Morton encoded/Z-order) cell id based on cell coordinates
                uint64_t cellID = bitInterlaceMagic((int)cell.x + i, (int)cell.y + j, (int)cell.z + k);
                // TODO: decide wether the if-statement below causes thread branching 
                if (cellStarts[cellID] == -1) {
                    continue; // cell is empty if it's start index is unchanged since last reset
                }
                // Iterate over all boids in neighbouring cell
                for (int l = cellStarts[cellID]; l <= cellEnds[cellID]; l++){
                    float distance = glm::distance(b.position, boids[l].position);
                    Boid neighbour = boids[l];
                    // exclude neighbours that are outside boid's scope 
                    float validNeighbour = (neighbour.position != b.position && distance < BOID_SCOPE);
                    neighbourCount += validNeighbour;
                    // Apply rules. Factor "validNeighbour" is zero for non-valid neighbours
                    alignment += validNeighbour * neighbour.velocity * 4.0f/(distance + 0.0000001f); // + 0.0001 is for avoiding divide by zero
                    separation += validNeighbour * (b.position - neighbour.position) * 1.0f/(float)(distance*distance + 0.0000001f); // + 0.0001 is for avoiding divide by zero
                    cohesion += validNeighbour * neighbour.position;
                }
            }
        }
    }
    // Divide by number of neighbours to get average values
    alignment = alignment * (1.0f / (neighbourCount + 1));
    // TODO: This is a debug quickfix, should not be an if here because it causes thread branching
    if( neighbourCount != 0){
	    cohesion = cohesion * (1.0f / (neighbourCount + 0.0000000001f)) - b.position; // We need 0.0000000001 here to avoid divide by zero
    }
    separation = separation * (1.0f / (neighbourCount + 0.0000000001f));
    
    /*Update Velocity*/
    glm::vec3 newVel = alignment + 50.0f*separation + 0.9f*cohesion;
    float speed = glm::clamp(length(newVel), MIN_SPEED, MAX_SPEED); // limit speed

	/* Update Velocity */
    newVelocities[i] = 0.01f*speed*glm::normalize(newVel);
    #ifdef DEBUG
    printf("Boid %d has %d neighbours\n", i, neighbourCount);
    #endif
} 

// Adds the new velocity value to the boids position, and copies the new velocity into the boid struct
// TODO: maybe we should place boid pos/vel in separate arrays, that way we don't have to copy the new velocities
// just swap pointers between two velocity arrays? 
__global__ void updatePosAndVel(Boid boids[], glm::vec3 newVelocities[], int nrBoids){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if(i >= nrBoids) return;
    glm::vec3 newPos = boids[i].position + newVelocities[i];
    // TODO: Right now we wrap the boids around a cube
    // TODO: This is just a quickfix. Just assigning MAX_COORD is not exactly accurate
    // Also, is a modulus operation possibly cheaper?
    newPos.x = newPos.x < CELL_SIZE ? MAX_COORD - CELL_SIZE : newPos.x;
    newPos.y = newPos.y < CELL_SIZE ? MAX_COORD - CELL_SIZE : newPos.y;
    newPos.z = newPos.z < CELL_SIZE ? MAX_COORD - CELL_SIZE : newPos.z;

	newPos.x = newPos.x > MAX_COORD - CELL_SIZE ? CELL_SIZE : newPos.x;
	newPos.y = newPos.y > MAX_COORD - CELL_SIZE ? CELL_SIZE : newPos.y;
	newPos.z = newPos.z > MAX_COORD - CELL_SIZE ? CELL_SIZE : newPos.z;

    boids[i].position = newPos;
    boids[i].velocity = newVelocities[i];
}

// Sets all the cell start/end indices to -1, so no old values is left
// TODO: only reset the ones that actually has had boids in it?
__global__ void resetCellRanges(int cellStarts[], int cellEnds[], int nrCells){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if(i < nrCells){
        cellStarts[i] = -1;
        cellEnds[i] = -1;
    }
}

// Stores the Morton code/Z-order value for each boid, based on the coordinates of the 
// cell which the boid currently is in
__global__ void calculateBoidHash(int n, uint64_t currentHashArray[], Boid b[]){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride){
        glm::vec3 cell = getCell(b[i].position);
        currentHashArray[i] = bitInterlaceMagic((int)cell.x, (int)cell.y, (int)cell.z);
    }
}

// After boid IDs are sorted the array with the actual boid structs are sorted accordingly with this function
__global__ void rearrangeBoids(int boidIDs[], Boid boids[], Boid boidsSorted[], int nrBoids){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if (i >= nrBoids) return;
    boidsSorted[i] = boids[boidIDs[i]]; // copy over boids to the boidsSorted array, which in the end will be sorted
}

void printCUDAError();

__global__ void prepareBoidRenderKernel(Boid* boids, glm::vec3* renderBoids, glm::mat4 projection, glm::mat4 view){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    int j = i*3;
    if(i >= NR_BOIDS) return;
    Boid b = boids[i];
    
    // one vector for each vertex
	const glm::vec3 p1(-1.0f, -1.0f, 0.0f);
	const glm::vec3 p2(0.0f, 1.0f, 0.0f);
    const glm::vec3 p3(1.0f, -1.0f, 0.0f);
    
    // create model matrix from agent position
    glm::mat4 model = glm::mat4(1.0f);
    model = glm::translate(model, b.position);
    glm::vec3 v = glm::vec3(b.velocity.z, 0, -b.velocity.x);
    float angle = acosf(b.velocity.y / glm::length(b.velocity)); // acosf is single precision == faster
    model = glm::rotate(model, angle, v);
    
    // transform each vertex and add them to array
    renderBoids[j] = view * model * glm::vec4(p1, 1.0f);
    renderBoids[j+1] = view * model * glm::vec4(p2, 1.0f);
    renderBoids[j+2] = view * model * glm::vec4(p3, 1.0f); 
}

void prepareBoidRender(Boid* boids, glm::vec3* renderBoids, glm::mat4 projection, glm::mat4 view){
    prepareBoidRenderKernel <<< numBlocksBoids, blockSize >>> (boids, renderBoids, projection, view);
}

void printCUDAInfo(){
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Memory Clock Rate (KHz): %d\n",
             prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n",
             prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
             2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}

void printCUDAError(){
    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

__host__ Boid** initBoidsOnGPU(Boid* boidsArr){
    printCUDAInfo();
    boids = boidsArr; // TODO: clean up all "boids" pointers

    // Allocate memory for the cell index arrays
    gpuErrchk( hipMallocManaged((void**)&cellStartIndex, sizeof(int) * NR_CELLS) );
    gpuErrchk( hipMallocManaged((void**)&cellEndIndex, sizeof(int) * NR_CELLS) );
    // Allocate memory for the temp storage of new velocities
    gpuErrchk( hipMallocManaged((void**)&newVelocities, sizeof(glm::vec3) * NR_BOIDS) );
    // Allocate memory for the boids
    gpuErrchk( hipMallocManaged((void**)&boids, sizeof(Boid) * NR_BOIDS) );
    gpuErrchk( hipMallocManaged((void**)&boidsSorted, sizeof(Boid) * NR_BOIDS) );
    // Allocate memory for the buffer arrays
    gpuErrchk( hipMallocManaged((void**)&boidCellIDs, sizeof(*boidCellIDs) * NR_BOIDS) );
    gpuErrchk( hipMallocManaged((void**)&boidCellIDsAlt, sizeof(*boidCellIDsAlt) * NR_BOIDS) );
    gpuErrchk( hipMallocManaged((void**)&boidIDs, sizeof(*boids) * NR_BOIDS) );
    gpuErrchk( hipMallocManaged((void**)&boidIDsAlt, sizeof(*boidIDsAlt) * NR_BOIDS) );

    boidCellIDsBuf = DoubleBuffer<uint64_t>(boidCellIDs, boidCellIDsAlt);
    boidIDsBuf = DoubleBuffer<int>(boidIDs, boidIDsAlt);
    return &boids; 
}

__host__ void deinitBoidsOnGPU(){
    // Free memory
    hipFree(cellStartIndex);
    hipFree(cellEndIndex);
    hipFree(boidCellIDsBuf.d_buffers[0]);
    hipFree(boidCellIDsBuf.d_buffers[1]);
    hipFree(boidIDsBuf.d_buffers[0]);
    hipFree(boidIDsBuf.d_buffers[1]);
    hipFree(newVelocities);
    hipFree(boids);
    hipFree(boidsSorted);
}


void cudaGraphicsGLRegisterBufferWrapper( struct hipGraphicsResource** positionsVBO_CUDA, unsigned int positionsVBO){
    // TODO: hipGraphicsRegisterFlagsWriteDiscard may be a useful flag here!
    hipGraphicsGLRegisterBuffer( positionsVBO_CUDA, positionsVBO, cudaGraphicsMapFlagsNone );
}

/* After calling this, you are free to execute CUDA kernels on the buffer! */ 
void mapBufferObjectCuda( struct hipGraphicsResource** positionsVBO_CUDA, size_t* num_bytes, glm::vec3** positions){
    hipGraphicsMapResources(1, positionsVBO_CUDA, 0);
    hipGraphicsResourceGetMappedPointer((void**)positions, num_bytes, *positionsVBO_CUDA);
}

void cudaGraphicsUnmapResourcesWrapper(struct hipGraphicsResource** positionsVBO_CUDA){
    hipGraphicsUnmapResources(1, positionsVBO_CUDA, 0);
}

void cudaGraphicsUnregisterResourceWrapper(struct hipGraphicsResource* positionsVBO_CUDA){
    hipGraphicsUnregisterResource(positionsVBO_CUDA);
}

void cudaSetDeviceWrapper(int n){
    hipSetDevice(n);
}

void step(){
    // Initialize boid id's
    initBoidIDs <<< numBlocksBoids, blockSize >>> (boidIDsBuf.Current(), NR_BOIDS);
    
    // Calculate cell IDs for every boid
    calculateBoidHash <<< numBlocksBoids, blockSize >>> (NR_BOIDS, boidCellIDsBuf.Current(), boids);
    
    // reset cell ranges
    resetCellRanges <<< numBlocksCells, blockSize >>> (cellStartIndex, cellEndIndex, NR_CELLS);
    
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    
    // Determine temporary storage need
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, boidCellIDsBuf, boidIDsBuf, NR_BOIDS);

    // Allocate temporary storage
    // TODO: hipMalloc is expensive, is it possible to do this particular allocation only once and reuse it? 
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, boidCellIDsBuf, boidIDsBuf, NR_BOIDS);
    
    hipFree(d_temp_storage);

    // Rearrange the actual boids based on the sorted boidIDs
    rearrangeBoids <<< numBlocksBoids, blockSize >>> (boidIDsBuf.Current(), boids, boidsSorted, NR_BOIDS);
    
    // Check were cellID changes occurs in the sorted boids array
    detectCellIndexChange <<< numBlocksBoids, blockSize >>> (cellStartIndex, cellEndIndex, boidCellIDsBuf.Current(), NR_BOIDS);

    // Update boid velocities based on the rules
    computeVelocities <<< numBlocksBoids, blockSize >>> (boidsSorted, cellStartIndex, cellEndIndex, boidCellIDsBuf.Current(), NR_BOIDS, newVelocities);
    
    // Copy boid velocities from temporary velocity storage to boid array
    updatePosAndVel <<< numBlocksBoids, blockSize >>> (boidsSorted, newVelocities, NR_BOIDS);

    // Swap the boids array pointer, so 'boids' now points to a sorted array
    hipDeviceSynchronize(); // TODO: is this call necessary?
    std::swap(boids, boidsSorted);
    
}