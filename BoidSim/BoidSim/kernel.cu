#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdint.h>
#include "boid.h"
#include "hipcub/hipcub.hpp"
#include "cub/device/device_radix_sort.cuh"

#include <stdio.h>
#include <algorithm>
#include "cub/util_allocator.cuh"
#include "cub/device/device_radix_sort.cuh"

/* 
Compile on Linux machines in NC:  
/chalmers/sw/sup64/cuda_toolkit-9.0.176.4/bin/nvcc --dont-use-profile -ldir /chalmers/sw/sup64/cuda_toolkit-9.0.176.4/nvvm/libdevice/ -I /chalmers/sw/sup64/cuda_toolkit-9.0.176.4/include -m64 -L /chalmers/sw/sup64/cuda_toolkit-9.0.176.4/lib64 ~/kernel.cu

You might need this before compiling: 
PATH=$PATH:/chalmers/sw/sup64/cuda_toolkit-9.0.176.4/nvvm/bin
PATH=$PATH:/chalmers/sw/sup64/cuda_toolkit-9.0.176.4/bin
*/ 

using namespace hipcub;

const int NR_BOIDS = 4;
const float MAX_COORD = 300.f; // World boundaries
const float CELL_SIZE = 10.f; // The world is divided in cubic cells  
const float BOID_SCOPE = 10.f; // this is how far boids look for neighbours. Should always be == CELL_SIZE ?

// Boid attributes
const float MAX_SPEED = 30.0f;
const float MIN_SPEED = 20.0f; // TODO 

// Calculate the maximum value of Morton encoded (Z-ordered) cell ids
#define shiftBitK(x, k) (int) ((x&(1<<k)) << k*2+2 | (x&(1<<k)) << k*2+1 | (x&(1<<k)) << k*2)
const int MAX_CELL_INDEX = (int) MAX_COORD/CELL_SIZE;
const int NR_CELLS = shiftBitK(MAX_CELL_INDEX, 10) 
                     |shiftBitK(MAX_CELL_INDEX, 9)
                     |shiftBitK(MAX_CELL_INDEX, 8)
                     |shiftBitK(MAX_CELL_INDEX, 7)
                     |shiftBitK(MAX_CELL_INDEX, 6)
                     |shiftBitK(MAX_CELL_INDEX, 5)
                     |shiftBitK(MAX_CELL_INDEX, 4)
                     |shiftBitK(MAX_CELL_INDEX, 3)
                     |shiftBitK(MAX_CELL_INDEX, 2)
                     |shiftBitK(MAX_CELL_INDEX, 1)
                     |shiftBitK(MAX_CELL_INDEX, 0);
// #define DEVICE_ENABLE

// A tempory storage for new velocities allows parallel processing of the boids velocities 
glm::vec3* newVelocities;

// These arrays hold the start and end indices for each cell which contains boids
int* cellStartIndex;
int* cellEndIndex;

// Get the cell based on the boids position
inline __device__ glm::vec3 getCell(glm::vec3 pos){
    return glm::floor(pos * (1.0f/CELL_SIZE));
}

// The boids are given an ID, which is the index it has in the intial boid array
__global__ void initBoidIDs(int BoidIDs[], int nrBoids){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x; 
    if(i >= nrBoids) return; 
    BoidIDs[i] = i;
}

// Spreads out bits with two zeroes in between
// TODO: find the source and give credit
__device__ uint64_t spreadOutByThree(uint64_t i){
    i = (i | (i << 16)) & 0x030000FF;
    i = (i | (i <<  8)) & 0x0300F00F;
    i = (i | (i <<  4)) & 0x030C30C3;
    i = (i | (i <<  2)) & 0x09249249;
    return i;
}


// Hash cell coords to morton code with "magic numbers"
__device__ uint64_t bitInterlaceMagic(int x, int y, int z){
    return spreadOutByThree((uint64_t)x) | (spreadOutByThree((uint64_t)y) << 1) | (spreadOutByThree((uint64_t)z) << 2);
}

// This function is used when scanning the sorted boids cell-ids to see were cells starts and ends 
// dummyHostIndex is only for debugging on machines without device enabled
__global__ void detectCellIndexChange(int cellStarts[], int cellEnds[], uint64_t cellIDs[], int nrBoids){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x; 
    if(i >= nrBoids) return; 
    
    int cellID = cellIDs[i];
    // TODO: determine if these if/else statements causes thread branching -> worse performance
    if(i == 0){ 
        // This is the case for the first element in the boid array 
        cellStarts[cellID] = i; 
    } else if (i == nrBoids - 1){ 
        // This is the case for the last element in the boid array
        cellEnds[cellID] = i;
    } else if (cellIDs[i] != cellIDs[i+1]){
        // A change in cell index was detected!
        cellStarts[cellIDs[i+1]] = i + 1;
        cellEnds[cellID] = i;
    }
}

// Update boid with index n
// WARNING! VERY MUCH TODO: RIGHT NOW IT CHECKS OUTSIDE WORLD BOUNDARIES FOR BOIDS
// IN EDGE CELLS, THIS WILL CAUSE ARRAY OUT OF-BOUNDS EXCEPTIONS
__global__ void computeVelocities(Boid boids[], int cellStarts[], int cellEnds[], uint64_t cellIDs[]
                                , int nrBoids, glm::vec3 newVelocities[]){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if(i >= nrBoids) return;
    int neighbourCount = 0;
    Boid b = boids[i]; // current boid whose neighbours we're checking
    // initialize default values for each rule
    glm::vec3 alignment = b.velocity;
	glm::vec3 separation = glm::vec3(0.0);
    glm::vec3 cohesion = glm::vec3(0.0);
    // Decide which cell current boid is in
    glm::vec3 cell = getCell(b.position);
    // Start checking all 27 neighbouring cells
    // TODO: find out a clever way to iterate over cells in order of the morton code to get 
    // more coherent memory accesses
    for(int i = -1; i <= 1; i++){
        for(int j = -1; j <= 1; j++){
            for(int k = -1; k <= 1; k++){
                // calculate the (Morton encoded/Z-order) cell id based on cell coordinates
                uint64_t cellID = bitInterlaceMagic((int)cell.x + i, (int)cell.y + j, (int)cell.z + k);
                // TODO: decide wether the if-statement below causes thread branching 
                if (cellStarts[cellID] == -1) {
                    continue; // cell is empty if it's start index is unchanged since last reset
                }
                // Iterate over all boids in neighbouring cell
                for (int l = cellStarts[cellID]; l <= cellEnds[cellID]; l++){
                    float distance = glm::distance(b.position, boids[l].position);
                    Boid neighbour = boids[l];
                    // exclude neighbours that are outside boid's scope 
                    float validNeighbour = (neighbour.position != b.position && distance < BOID_SCOPE);
                    neighbourCount += validNeighbour;
                    // Apply rules. Factor "validNeighbour" is zero for non-valid neighbours
                    alignment += validNeighbour * neighbour.velocity * 4.0f/(distance + 0.0000001f); // + 0.0001 is for avoiding divide by zero
                    separation += validNeighbour * (b.position - neighbour.position) * 1.0f/(float)(pow(distance,2) + 0.0000001f); // + 0.0001 is for avoiding divide by zero
                    cohesion += validNeighbour * neighbour.position;
                }
            }
        }
    }
    // Divide by number of neighbours to get average values
    alignment = alignment * (1.0f / (neighbourCount + 1));
    // TODO: This is a debug quickfix, should not be an if here because it causes thread branching
    if( neighbourCount != 0){
	    cohesion = cohesion * (1.0f / (neighbourCount + 0.0000000001f)) - b.position; // We need 0.0000000001 here to avoid divide by zero
    }
    separation = separation * (1.0f / (neighbourCount + 0.0000000001f));
    printf("Boid %d has cohesion %f, %f, %f \n", i, cohesion.x, cohesion.y, cohesion.z);
    printf("Boid %d has separation %f, %f, %f \n", i, separation.x, separation.y, separation.z);
    printf("Boid %d has alignment %f, %f, %f \n", i, alignment.x, alignment.y, alignment.z);

    /*Update Velocity*/
    glm::vec3 newVel = alignment + 50.0f*separation + 0.9f*cohesion;
    // std::cout << "Boid " << n << " has newVel " << newVel.x << ", " << newVel.y << ", " << newVel.z << std::endl;
    float speed = glm::clamp(length(newVel), MIN_SPEED, MAX_SPEED); // limit speed

	/* Update Velocity */
    newVelocities[i] = speed*glm::normalize(newVel);
    
    printf("Boid %d has %d neighbours\n", i, neighbourCount);
} 

// Adds the new velocity value to the boids position, and copies the new velocity into the boid struct
// TODO: maybe we should place boid pos/vel in separate arrays, that way we don't have to copy the new velocities
// just swap pointers between two velocity arrays? 
__global__ void updatePosAndVel(Boid boids[], glm::vec3 newVelocities[], int nrBoids){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if(i >= nrBoids) return;
    boids[i].position += newVelocities[i];
    printf("Updating boid %d with new velocity: %f, %f, %f \n", i, newVelocities[i].x, newVelocities[i].y,  newVelocities[i].z);
    boids[i].velocity = newVelocities[i];
}

// Sets all the cell start/end indices to -1, so no old values is left
// TODO: only reset the onces that actually has had boids in it
__global__ void resetCellRanges(int cellStarts[], int cellEnds[], int nrCells){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if(i < nrCells){
        cellStarts[i] = -1;
        cellEnds[i] = -1;
    }
}

// Stores the Morton code/Z-order value for each boid, based on the coordinates of the 
// cell which the boid currently is in
__global__ void calculateBoidHash(int n, uint64_t currentHashArray[], Boid b[]){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride){
        glm::vec3 cell = getCell(b[i].position);
        printf("Boid nr %d has pos %f, %f, %f", i, b[i].position.x , b[i].position.y , b[i].position.z);
        printf(" is in cell %f, %f, %f \n", cell.x, cell.y, cell.z);
        currentHashArray[i] = bitInterlaceMagic((int)cell.x, (int)cell.y, (int)cell.z);
        printf("has morton code: %ld \n", (long)currentHashArray[i]);
    }
}

// After boid IDs are sorted the array with the actual boid structs are sorted accordingly with this function
__global__ void rearrangeBoids(int boidIDs[], Boid boids[], Boid boidsSorted[], int nrBoids){
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if (i >= nrBoids) return;
    boidsSorted[i] = boids[boidIDs[i]]; // copy over boids to the boidsSorted array, which in the end will be sorted
}

void printCUDAInfo(){
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Memory Clock Rate (KHz): %d\n",
             prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n",
             prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
             2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}

void printCUDAError(){
    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

int main(int argc, char** argv){

    printCUDAInfo();
    // These arrays hold the boids
    Boid* boids = NULL;
    Boid* boidsSorted = NULL;
    // These arrays hold the (Z-order/morton encoded) cell ids
    uint64_t* boidCellIDs = NULL;
    uint64_t* boidCellIDsAlt = NULL;
    // Array with all the boids. boidsSorted is a alternate array needed for the radixSort
    int* boidIDs = NULL;
    int* boidIDsAlt = NULL;

    // Allocate memory for the cell index arrays
    hipMallocManaged((void**)&cellStartIndex, sizeof(int) * NR_CELLS);
    hipMallocManaged((void**)&cellEndIndex, sizeof(int) * NR_CELLS);
    // Allocate memory for the temp storage of new velocities
    hipMallocManaged((void**)&newVelocities, sizeof(glm::vec3) * NR_BOIDS);
    // Allocate memory for the boids
    hipMallocManaged((void**)&boids, sizeof(Boid) * NR_BOIDS);
    hipMallocManaged((void**)&boidsSorted, sizeof(Boid) * NR_BOIDS);
    // Allocate memory for the buffer arrays
    hipMallocManaged((void**)&boidCellIDs, sizeof(*boidCellIDs) * NR_BOIDS);
    hipMallocManaged((void**)&boidCellIDsAlt, sizeof(*boidCellIDsAlt) * NR_BOIDS);
    hipMallocManaged((void**)&boidIDs, sizeof(*boids) * NR_BOIDS);
    hipMallocManaged((void**)&boidIDsAlt, sizeof(*boidIDsAlt) * NR_BOIDS);

    DoubleBuffer<uint64_t> boidCellIDsBuf(boidCellIDs, boidCellIDsAlt);
    DoubleBuffer<int> boidIDsBuf(boidIDs, boidIDsAlt);

    // Some test values for the boids
    boids[0].position = glm::vec3(20,20,20);
    boids[1].position = glm::vec3(20,20,25);
    boids[2].position = glm::vec3(20,40,100);
    boids[3].position = glm::vec3(20,40,80);

    boids[0].velocity = glm::vec3(0,0,0);
    boids[1].velocity = glm::vec3(0,0,0);
    boids[2].velocity = glm::vec3(0,0,1);
    boids[3].velocity = glm::vec3(0,0,1);

    int blockSize = 256;
    int numBlocksBoids = (NR_BOIDS + blockSize - 1) / blockSize;
    int numBlocksCells = (NR_CELLS + blockSize - 1) / blockSize;

    // Initialize boid id's
    initBoidIDs <<< numBlocksBoids, blockSize >>> (boidIDs, NR_BOIDS);

    // Calculate cell IDs for every boid
    calculateBoidHash <<< numBlocksBoids, blockSize >>> (NR_BOIDS, boidCellIDsBuf.Current(), boids);
    hipDeviceSynchronize();

    // reset cell ranges
    resetCellRanges <<< numBlocksCells, blockSize >>> (cellStartIndex, cellEndIndex, NR_CELLS);
    hipDeviceSynchronize();

    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    
    // Determine temporary storage need
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, boidCellIDsBuf, boidIDsBuf, NR_BOIDS);
    hipDeviceSynchronize();

    // Allocate temporary storage
    hipMallocManaged(&d_temp_storage, temp_storage_bytes);
    hipDeviceSynchronize();
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, boidCellIDsBuf, boidIDsBuf, NR_BOIDS);
    printCUDAError();
    hipDeviceSynchronize();

    // Rearrange the actual boids based on the sorted boidIDs
    rearrangeBoids <<< numBlocksBoids, blockSize >>> (boidIDsBuf.Current(), boids, boidsSorted, NR_BOIDS);
    // Swap the array pointers

    // Check were cellID changes occurs in the sorted boids array
    detectCellIndexChange <<< numBlocksBoids, blockSize >>> (cellStartIndex, cellEndIndex, boidCellIDsBuf.Current(), NR_BOIDS);
    hipDeviceSynchronize();

    // Update boid velocities based on the rules
    computeVelocities <<< numBlocksBoids, blockSize >>> (boidsSorted, cellStartIndex, cellEndIndex, boidCellIDsBuf.Current(), NR_BOIDS, newVelocities);
    hipDeviceSynchronize();
    printCUDAError();

    // Copy boid velocities from temporary velocity storage to boid array
    updatePosAndVel <<< numBlocksBoids, blockSize >>> (boidsSorted, newVelocities, NR_BOIDS);
    hipDeviceSynchronize();

    // Swap the boids array pointer, so 'boids' now points to a sorted array
    std::swap(boids, boidsSorted);

    // Print results
    std::cout << "(cellID, z-position, cell start index, cell end index)\n";
    for(int i = 0; i < NR_BOIDS; i++){
        uint64_t cellID = boidCellIDsBuf.Current()[i];
        std::cout << "(" << cellID << ", " << boidsSorted[i].position.z;
        std::cout << ", " << cellStartIndex[cellID] << ", " << cellEndIndex[cellID] << ")" << std::endl;
    }
    // Free memory
    hipFree(cellStartIndex);
    hipFree(&d_temp_storage);
    hipFree(cellEndIndex);
    hipFree(boidCellIDsBuf.d_buffers[0]);
    hipFree(boidCellIDsBuf.d_buffers[1]);
    hipFree(boidIDsBuf.d_buffers[0]);
    hipFree(boidIDsBuf.d_buffers[1]);
    hipFree(newVelocities);
    hipFree(boids);
    hipFree(boidsSorted);
}